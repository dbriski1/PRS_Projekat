#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "graph.cuh"
#include <stdio.h>

__global__ void kernel_cuda_simple(
	int num_vertices,
	int* v_adj_length,
	int* v_adj_list,
	int* v_adj_begin,
	int* result,
	bool* still_running
	)
 {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	

	for (int v = 0; v < num_vertices; v += num_threads)
	{
		int vertex = v + tid;
		//printf("%d ", num_vertices);
		if (vertex < num_vertices)
		{
			
			
			//printf("Velicina adjacency matrice za vertex %d: %d\n", vertex, v_adj_length[vertex]);
			for (int n = 0; n < v_adj_length[vertex]; n++)
			{

				int neighbor = v_adj_list[v_adj_begin[vertex] + n];
				//printf("Susjed cvora %d je: %d\n",vertex, neighbor);
					if (result[neighbor] > result[vertex] + 1)
					{
						printf("%d %d\n", result[neighbor], result[vertex] + 1);
						result[neighbor] = result[vertex] + 1;
						* still_running = 1;
						printf("Rezultat za izvorni cvor %d je %d, a za ciljni cvor %d je %d\n", vertex, result[vertex], neighbor, result[neighbor]);
					}
				
			}
		}
	}
	printf("\n");
}

 void run(Graph G, std::vector<int> &resultVector)
 { 
	 
	 int N = G.numVertices;
	 
	 int * d_adj_length, * d_adj_list, * d_adj_begin, *d_result;
	 bool* d_still_running;
	 //bool* h_test = new bool(true);
	 

	 /*hipMalloc(&h_test, N * sizeof(bool));
	 for (int k = 0; k < N; k++) h_test[k] = true;*/

	 bool* k_still_running = new bool(true);
	 //hipMalloc(&k_still_running, N * sizeof(bool));

	 bool* still_running = (bool*)malloc(1 * sizeof(bool));;
	 //hipMalloc(&still_running, N * sizeof(bool));


	 int BLOCKS = 1;
	 dim3 THREADS(N, 1, 1);

	 int* adj_length = &(*G.edgesSize.begin());
	 int* adj_list = &(*G.adjacencyList.begin());
	 int* adj_begin = &(*G.edgesOffset.begin());
	 int* result = &(*resultVector.begin());

	 hipMalloc(&d_still_running, 1 * sizeof(bool));
	 hipMalloc(&d_adj_length, G.edgesSize.size() * sizeof(int));
	 hipMalloc(&d_adj_list, G.adjacencyList.size() * sizeof(int));
	 hipMalloc(&d_adj_begin, G.edgesOffset.size() * sizeof(int));
	 hipMalloc(&d_result, N * sizeof(int));

	 

	 //printf("%d ", adj_begin[0]);

	 while (*still_running) {
		 {
			 *still_running = false;
			 *k_still_running = false;
			 hipMemcpy(d_still_running, still_running, sizeof(bool) * 1, hipMemcpyHostToDevice);
			 hipMemcpy(d_adj_length, adj_length, G.edgesSize.size() * sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(d_adj_list, adj_list, G.adjacencyList.size() * sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(d_adj_begin, adj_begin, G.edgesOffset.size() * sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(d_result, result, N * sizeof(int), hipMemcpyHostToDevice);
			 //hipMemcpy(d_y, y, N * sizeof(int), hipMemcpyHostToDevice);
			 
			 for (int i = 0; i < N; i++) {
				 printf("%d ", result[i]);
			 }
			 printf("\n");

			 kernel_cuda_simple <<<BLOCKS, THREADS>>> (N, d_adj_length, d_adj_list, d_adj_begin, d_result, d_still_running);
			 //hipLaunchKernel(reinterpret_cast<const void*>(kernel_cuda_simple), G.numVertices / 1024 + 1, 1024, 1, args, 0, 0);
			 hipMemcpy(still_running, d_still_running, sizeof(bool) * 1, hipMemcpyDeviceToHost);
			 hipMemcpy(adj_length, d_adj_length, G.edgesSize.size() * sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(adj_list, d_adj_list, G.adjacencyList.size() * sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(adj_begin, d_adj_begin, G.edgesOffset.size() * sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(result, d_result, N * sizeof(int), hipMemcpyDeviceToHost);
		 }
	 }
	
	hipDeviceSynchronize();
	hipFree(still_running);
	hipFree(k_still_running);

 }

 int main() {
	 Graph G;
	 //int startVertex = atoi(argv[1]);
	 readGraph(G);

	 /*std::vector<int> distance;
	 std::vector<int> parent;
	 std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
	 std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());*/

	 printf("Number of vertices %d\n", G.numVertices);
	 printf("Number of edges %d\n\n", G.numEdges);


	 //vectors for results
	 std::vector<int> resultVector(G.numVertices, std::numeric_limits<int>::max());
	 resultVector[0] = 0;


	 run(G, resultVector);
	 for (int i = 0; i < resultVector.size(); i++) {
		 printf("%d  ", resultVector[i]);
	 }
	 return 0;
 }