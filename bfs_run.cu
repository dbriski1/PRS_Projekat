#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "graph.cuh"
#include <stdio.h>
#include <chrono>


//  Vertex-centric Implementation Explicit Iteration Counter
__global__ void kernel_cuda_simple(
	long int num_vertices,
	long int level,
	long int* v_adj_length,
	long int* v_adj_list,
	long int* v_adj_begin,
	long int* result,
	bool* still_running
	)
 {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	//printf("Broj tid: %d\n", tid);

	for (long int v = 0; v < num_vertices; v += num_threads)
	{
		long int vertex = v + tid;
		//printf("Trenutni thread je %d, a vertex je %d\n", tid, vertex);
		if (result[vertex] == level && vertex < num_vertices)
		{
			//printf("Usao u vertex broj: %d\n", vertex);
			
			//printf("Velicina adjacency matrice za vertex %d: %d\n", vertex, v_adj_length[vertex]);
			for (long int n = 0; n < v_adj_length[vertex]; n++)
			{

				long int neighbor = v_adj_list[v_adj_begin[vertex] + n];
				//printf("Susjed cvora %d je: %d\n",vertex, neighbor);
					if (result[neighbor] > result[vertex] + 1)
					{
						//printf("%d %d\n", result[neighbor], result[vertex] + 1);
						result[neighbor] = result[vertex] + 1;
						* still_running = 1;
						//printf("Rezultat za izvorni cvor %d je %d, a za ciljni cvor %d je %d\n", vertex, result[vertex], neighbor, result[neighbor]);
					}
				
			}
		}
	}
	//printf("\n");
}

__global__ void kernel_cuda_optimized(
	long int num_vertices,
	long int level,
	long int* v_adj_length,
	long int* v_adj_list,
	long int* v_adj_begin,
	long int* result,
	bool* still_running
)
{
	long int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num_vertices && result[tid] == level) {
		long int u = tid;
		for (long int i = v_adj_begin[u]; i < v_adj_begin[u] + v_adj_length[u]; i++) {
			long int v = v_adj_list[i];
			if (level + 1 < result[v]) {
				result[v] = level + 1;
				/*d_parent[v] = i;*/
				*still_running = 1;
			}
		}
	}
	//printf("\n");
}

 void run_simple(Graph G, std::vector<long int> &resultVector)
 { 
	 
	 
	 long int N = G.numVertices;
	 
	 long int * d_adj_length, * d_adj_list, * d_adj_begin, *d_result, d_level;
	 bool* d_still_running;
	 
	 //bool* h_test = new bool(true);
	 

	 /*hipMalloc(&h_test, N * sizeof(bool));
	 for (int k = 0; k < N; k++) h_test[k] = true;*/

	 bool* k_still_running = new bool(true);
	 //hipMalloc(&k_still_running, N * sizeof(bool));

	 bool* still_running = (bool*)malloc(1 * sizeof(bool));;
	 //hipMalloc(&still_running, N * sizeof(bool));


	 int BLOCKS = 8;
	 dim3 THREADS(128, 1, 1);

	 long int* adj_length = &(*G.edgesSize.begin());
	 long int* adj_list = &(*G.adjacencyList.begin());
	 long int* adj_begin = &(*G.edgesOffset.begin());
	 long int* result = &(*resultVector.begin());

	 hipMalloc(&d_still_running, 1 * sizeof(bool));
	 hipMalloc(&d_adj_length, G.edgesSize.size() * sizeof(long int));
	 hipMalloc(&d_adj_list, G.adjacencyList.size() * sizeof(long int));
	 hipMalloc(&d_adj_begin, G.edgesOffset.size() * sizeof(long int));
	 hipMalloc(&d_result, N * sizeof(long int));
	 

	 /*printf("Velicina edges size: %d\n", G.edgesSize.size());
	 printf("Velicina adjacency list: %d\n", G.adjacencyList.size());
	 printf("Velicina edges offset: %d\n", G.edgesOffset.size());

	 printf("Starting simple parallel bfs.\n");*/
	 
	 long int level = 0;
	 hipMemcpy(d_adj_length, adj_length, G.edgesSize.size() * sizeof(long int), hipMemcpyHostToDevice);
	 hipMemcpy(d_adj_list, adj_list, G.adjacencyList.size() * sizeof(long int), hipMemcpyHostToDevice);
	 hipMemcpy(d_adj_begin, adj_begin, G.edgesOffset.size() * sizeof(long int), hipMemcpyHostToDevice);
	 hipMemcpy(d_result, result, N * sizeof(long int), hipMemcpyHostToDevice);

	 auto start = std::chrono::steady_clock::now();
	 while (*still_running) {
		 {
			 *still_running = false;
			 hipMemcpy(d_still_running, still_running, sizeof(bool) * 1, hipMemcpyHostToDevice);
			 

			 kernel_cuda_simple <<<BLOCKS, THREADS>>> (N, level, d_adj_length, d_adj_list, d_adj_begin, d_result, d_still_running);
			 hipMemcpy(still_running, d_still_running, sizeof(bool) * 1, hipMemcpyDeviceToHost);

			 level++;
			 
		 }
	 }
	 hipMemcpy(result, d_result, N * sizeof(long int), hipMemcpyDeviceToHost);


	auto end = std::chrono::steady_clock::now();
	long duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	hipDeviceSynchronize();
	hipFree(still_running);
	hipFree(k_still_running);
	hipFree(d_adj_length);
	hipFree(d_adj_list);
	hipFree(d_adj_begin);
	hipFree(d_result);
	printf("Elapsed time in microseconds for simple BFS: %li ms.\n", duration);
 }

 void run_optimized(Graph G, std::vector<long int>& resultVector)
 {


	 long int N = G.numVertices;

	 long int* d_adj_length, * d_adj_list, * d_adj_begin, * d_result, d_level;
	 bool* d_still_running;

	 //bool* h_test = new bool(true);


	 /*hipMalloc(&h_test, N * sizeof(bool));
	 for (int k = 0; k < N; k++) h_test[k] = true;*/

	 bool* k_still_running = new bool(true);
	 //hipMalloc(&k_still_running, N * sizeof(bool));

	 bool* still_running = (bool*)malloc(1 * sizeof(bool));;
	 //hipMalloc(&still_running, N * sizeof(bool));


	 int BLOCKS = 8;
	 dim3 THREADS(128, 1, 1);

	 long int* adj_length = &(*G.edgesSize.begin());
	 long int* adj_list = &(*G.adjacencyList.begin());
	 long int* adj_begin = &(*G.edgesOffset.begin());
	 long int* result = &(*resultVector.begin());

	 hipMalloc(&d_still_running, 1 * sizeof(bool));
	 hipMalloc(&d_adj_length, G.edgesSize.size() * sizeof(long int));
	 hipMalloc(&d_adj_list, G.adjacencyList.size() * sizeof(long int));
	 hipMalloc(&d_adj_begin, G.edgesOffset.size() * sizeof(long int));
	 hipMalloc(&d_result, N * sizeof(long int));


	 /*printf("Velicina edges size: %d\n", G.edgesSize.size());
	 printf("Velicina adjacency list: %d\n", G.adjacencyList.size());
	 printf("Velicina edges offset: %d\n", G.edgesOffset.size());

	 printf("Starting simple parallel bfs.\n");*/

	 long int level = 0;
	 hipMemcpy(d_adj_length, adj_length, G.edgesSize.size() * sizeof(long int), hipMemcpyHostToDevice);
	 hipMemcpy(d_adj_list, adj_list, G.adjacencyList.size() * sizeof(long int), hipMemcpyHostToDevice);
	 hipMemcpy(d_adj_begin, adj_begin, G.edgesOffset.size() * sizeof(long int), hipMemcpyHostToDevice);
	 hipMemcpy(d_result, result, N * sizeof(long int), hipMemcpyHostToDevice);

	 auto start = std::chrono::steady_clock::now();
	 while (*still_running) {
		 {
			 *still_running = false;
			 hipMemcpy(d_still_running, still_running, sizeof(bool) * 1, hipMemcpyHostToDevice);


			 kernel_cuda_optimized << <BLOCKS, THREADS >> > (N, level, d_adj_length, d_adj_list, d_adj_begin, d_result, d_still_running);
			 hipMemcpy(still_running, d_still_running, sizeof(bool) * 1, hipMemcpyDeviceToHost);

			 level++;

		 }
	 }
	 hipMemcpy(result, d_result, N * sizeof(long int), hipMemcpyDeviceToHost);


	 auto end = std::chrono::steady_clock::now();
	 long duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	 hipDeviceSynchronize();
	 hipFree(still_running);
	 hipFree(k_still_running);
	 hipFree(d_adj_length);
	 hipFree(d_adj_list);
	 hipFree(d_adj_begin);
	 hipFree(d_result);
	 printf("Elapsed time in microseconds for optimized BFS: %li ms.\n", duration);
 }


 int main() {
	 Graph G;
	 readGraph(G);

	 printf("Number of vertices %d\n", G.numVertices);
	 printf("Number of edges %d\n\n", G.numEdges);

	 std::vector<long> resultVector_simple(G.numVertices, std::numeric_limits<long>::max());
	 resultVector_simple[0] = 0;


	 run_simple(G, resultVector_simple);
	 for (long i = 0; i < resultVector_simple.size(); i++) {
		 //printf("%d %d  \n",i, resultVector_simple[i]);
	 }
	 // Result for p2p-Gnutella08.txt is around 2300 microseconds

	 std::vector<long> resultVector_optimized(G.numVertices, std::numeric_limits<long>::max());
	 resultVector_optimized[0] = 0;


	 run_optimized(G, resultVector_optimized);
	 for (long i = 0; i < resultVector_optimized.size(); i++) {
		 //printf("%d %d  \n", i, resultVector_optimized[i]);
	 }
	 // Result for p2p-Gnutella08.txt is around 1200 microseconds

	 return 0;
 }